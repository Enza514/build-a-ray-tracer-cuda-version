﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>

#include "common.cuh"
#include "vec3.cuh"
#include "colour.cuh"

int main()
{
    
    // Image Size

    int image_width = 1024;
    int image_height = 1024;

    // Render
    std::ofstream output("image.ppm");
    output << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = 0; j < image_height; j++) {
        std::clog << "\rScanlines remaining: " << (image_height - j) << ' ' << std::flush;
        for (int i = 0; i < image_width; i++) {
            auto pixel_colour = colour(double(i)/(image_width-1), double(j)/(image_height-1), 0);
            write_colour(output, pixel_colour);
        }
    }
    std::clog << "\rDone.                 \n";

    return 0;
}
